#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include "nn.h"
#include "nn_gpu.h"
/* -*- mode: c -*- */

#include <float.h>
#include "gpu_tree.h"

__global__ void init_kernel(void) {

}

__global__ void nearest_neighbor_search (gpu_tree gpu_tree, gpu_point *d_training_points, int n_training_points,
																				 gpu_point *d_search_points, int n_search_points) 
{

 	float search_points_coord[DIM];
	int closest;
	float closestDist;

#ifdef TRACK_TRAVERSALS
	int numNodesTraversed;
#endif

	int i, j, pidx;
	
	int cur_node_index, prev_node_index;
	__shared__ int SP[NUM_WARPS_PER_BLOCK];
#define sp SP[WARP_IDX]
	
	__shared__ int stk[NUM_WARPS_PER_BLOCK][64];
	__shared__ extern int tmp_shr[];
	int stk_top;

	bool cond, status;
    bool opt1, opt2;
	int critical;
	unsigned int vote_left;
	unsigned int vote_right;
	unsigned int num_left;
	unsigned int num_right;

	gpu_tree_node_0 cur_node0;
	gpu_tree_node_2 cur_node2;
	gpu_tree_node_3 cur_node3;

	float dist=0.0;
	float boxdist=0.0;
	float sum=0.0;
	float boxsum=0.0;
	float center=0.0;
	int id = 0;

#include "nn_kernel_macros.inc"

	for (pidx = blockIdx.x * blockDim.x + threadIdx.x; pidx < n_search_points; pidx += blockDim.x * gridDim.x)
    {
		for(j = 0; j < DIM; j++) {
			search_points_coord[j] = d_search_points[pidx].coord[j];
		}

		closest = d_search_points[pidx].closest;
		closestDist = d_search_points[pidx].closestDist;
#ifdef TRACK_TRAVERSALS
		numNodesTraversed = 0; //d_search_points[pidx].numNodesTraversed;
#endif

		cur_node_index = 0;
		STACK_INIT ();
		STACK_NODE = 0;
		status = 1;
		critical = 63;
		cond = 1;

		while(sp >= 1) {
			cur_node_index = STACK_NODE;

//			if (pidx == 0) {
//				printf("status = %d, critical = %d, sp = %d, and cur_node_index = %d\n", status, critical, sp, cur_node_index);
//			}

			if (status == 0 && critical >= sp) {
				status = 1;
			}
			STACK_POP();

			if (status == 1) {
#ifdef TRACK_TRAVERSALS
			numNodesTraversed++;
#endif
//                critical = sp - 1;
//                if (pidx == 0) {
//                    printf("cur_node_index = %d\n", cur_node_index);
//                }
				//cur_node1 = gpu_tree.nodes1[cur_node_index];
				// inlined function can_correlate
				dist=0.0;
				boxdist=0.0;
				sum=0.0;
				boxsum=0.0;
				center=0.0;

				for(i = 0; i < DIM; i++) {
					float max = gpu_tree.nodes1[cur_node_index].items[i].max;
					float min = gpu_tree.nodes1[cur_node_index].items[i].min;
				    center = (max + min) / 2;
					boxdist = (max - min) / 2;
					dist = search_points_coord[i] - center;
					sum += dist * dist;
					boxsum += boxdist * boxdist;
				}

				cond = (sqrt(sum) - sqrt(boxsum) < sqrt(closestDist));
                critical = sp;
				if(!__any(cond)) {
					continue;
				}

//                critical = sp - 1;
				if (!cond) {
					status = 0;
//                    critical = sp - 1;
				} else {
					cur_node0 = gpu_tree.nodes0[cur_node_index];
					if(cur_node0.items.axis == DIM) {
						cur_node3 = gpu_tree.nodes3[cur_node_index];
						for(i = 0; i < MAX_POINTS_IN_CELL; i++) {
							if(cur_node3.points[i] >= 0) {
								// update closest...
								float dist = 0.0;
								float t;

								for(j = 0; j < DIM; j++) {
									t = (d_training_points[cur_node3.points[i]].coord[j] - search_points_coord[j]);
									dist += t*t;
								}

								if(dist <= closestDist) {
									closest = cur_node3.points[i];
									closestDist = dist;
								}
							}
						}

					} else {
						cur_node2 = gpu_tree.nodes2[cur_node_index];
						opt1 = search_points_coord[cur_node0.items.axis] < cur_node0.items.splitval;
						opt2 = search_points_coord[cur_node0.items.axis] >= cur_node0.items.splitval;
						vote_left = __ballot(opt1);
						vote_right = __ballot(opt2);
						num_left = __popc(vote_left);
						num_right = __popc(vote_right);
						// majority vote
						if (num_left > num_right) {
							if(RIGHT != NULL_NODE) { STACK_PUSH(RIGHT); }
							if(LEFT != NULL_NODE) { STACK_PUSH(LEFT); }
						} else {
							if(LEFT != NULL_NODE) { STACK_PUSH(LEFT); }
							if(RIGHT != NULL_NODE) { STACK_PUSH(RIGHT); }
						}
					}
				}
			}
		}

		d_search_points[pidx].closest = closest;
		d_search_points[pidx].closestDist = closestDist;
#ifdef TRACK_TRAVERSALS
		d_search_points[pidx].numNodesTraversed = numNodesTraversed;
#endif

	}
}
 
int sort_flag = 0;
int verbose_flag = 0;
int check_flag = 0;
int ratio_flag = 0;
int warp_flag = 0;

Point *training_points;
KDCell *root;
Point *search_points;

int npoints;
int nsearchpoints;
char *input_file;

static inline float distance_axis(Point *a, Point *b, int axis);
static inline float distance(Point *a, Point *b);

TIME_INIT(runtime);
TIME_INIT(construct_tree);
TIME_INIT(gpu_build_tree);
TIME_INIT(init_kernel);
TIME_INIT(gpu_copy_to);
TIME_INIT(gpu_copy_from);
TIME_INIT(kernel);
TIME_INIT(sort);
TIME_INIT(traversal_time);


static int leaves = 0;
static int max_depth = 0;
static int number_of_nodes = 0;
void PrintTree(KDCell * root, int depth, int id){
	if (!root) {
		return;
	}
	number_of_nodes ++;
	if (depth > max_depth) {
		max_depth = depth;
	}
	for (int i = 0; i < depth; i ++) {
		printf(" ");
	}
	printf("NODE = %d, Depth = %d, axis = %d, ", id, depth, root->axis);
	if (root->axis == DIM) {
//	if (root->left == NULL && root->right == NULL) {
		printf("Type: = LEAFNODE!\n");
		leaves ++;
	} else {
		printf("Type = non\n");
	}
	if (root->left) {
		PrintTree(root->left, depth + 1, id * 2);
	}
	if (root->right) {
		PrintTree(root->right, depth + 1, id * 2 + 1);
	}
}

int main(int argc, char **argv) {

	int correct_cnt, i, j;
	unsigned long long sum_nodes_traversed;
	float correct_rate;
	
	struct thread_args *args;
	pthread_t *threads;

	read_input(argc, argv);
	printf("configuration: sort_flag=%d verbose_flag=%d check_flag=%d DIM = %d npoints = %d nsearchpoints = %d\n", sort_flag, verbose_flag, check_flag, DIM, npoints, nsearchpoints);

	TIME_START(runtime);
	TIME_START(construct_tree);

	if(sort_flag) {
		TIME_START(sort);
		sort_points(search_points, 0, nsearchpoints - 1, 0);
		TIME_END(sort);
	}

	root = construct_tree(training_points, 0, npoints - 1, 0, 1);
	
//	PrintTree(root, 0, 1);
//	printf("Number of leaves is %d. Max depth is %d, Number of total nodes is %d.\n", leaves, max_depth, number_of_nodes);
//	exit(0);

	TIME_END(construct_tree);
	TIME_START(traversal_time);
	
	TIME_START(gpu_build_tree);
	gpu_tree *h_tree = gpu_transform_tree(root);
	gpu_point *h_training_points = gpu_transform_points(training_points, npoints);
	gpu_point *h_search_points = gpu_transform_points(search_points, nsearchpoints);
	TIME_END(gpu_build_tree);

	TIME_START(init_kernel);
	init_kernel<<<1,1>>>();
	TIME_END(init_kernel);

	TIME_START(gpu_copy_to);
	gpu_tree *d_tree = gpu_copy_to_dev(h_tree);
	gpu_free_tree_host(h_tree);

	gpu_point *d_training_points = gpu_copy_points_to_dev(h_training_points, npoints);
	gpu_free_points_host(h_training_points);
 
	gpu_point *d_search_points = gpu_copy_points_to_dev(h_search_points, nsearchpoints);
	TIME_END(gpu_copy_to);

	dim3 grid(NUM_THREAD_BLOCKS, 1, 1);
	dim3 block(NUM_THREADS_PER_BLOCK, 1, 1);
	TIME_START(kernel);
	nearest_neighbor_search<<<grid, block, 3840>>>(*d_tree, d_training_points, npoints, d_search_points, nsearchpoints);

	hipError_t err = hipDeviceSynchronize();
	TIME_END(kernel);

	if(err != hipSuccess) {
		fprintf(stderr,"Kernel failed with error: %s\n", hipGetErrorString(err));
		exit(1);
	}
	TIME_START(gpu_copy_from);

	// copy back into Points from the 
	gpu_copy_points_to_host(d_search_points, h_search_points, search_points, nsearchpoints);
	
	// free device data
	gpu_free_points_host(h_search_points);
	gpu_free_points_dev(d_search_points);
	gpu_free_points_dev(d_training_points);
	gpu_free_tree_dev(d_tree);
 	
	TIME_END(gpu_copy_from);
	TIME_END(traversal_time);
	TIME_END(runtime);

	correct_cnt = 0;
	for(i = 0; i < nsearchpoints; i++) {
			if(search_points[i].closest >= 0) {
				if (training_points[search_points[i].closest].label == search_points[i].label) {
					correct_cnt++;
				}
		}
	}
	
	correct_rate = (float) correct_cnt / nsearchpoints;
	printf("correct rate: %.4f\n", correct_rate);

	#ifdef TRACK_TRAVERSALS
	sum_nodes_traversed = 0;
    int maximum = 0, all = 0;
    unsigned long long maximum_sum = 0, all_sum = 0;
	for (i = 0; i < nsearchpoints + (nsearchpoints % 32); i+=32) {
		int na =search_points[i].numNodesTraversed;
//        printf("nodes warp %d: %d\n", i/32, na);
        sum_nodes_traversed += search_points[i].numNodesTraversed;

        if (warp_flag) {
            maximum = na;
            all = na;
            for(j = i + 1; j < i + 32 && j < nsearchpoints; j++) {
		    	sum_nodes_traversed += search_points[j].numNodesTraversed;
	    		if(search_points[j].numNodesTraversed)
    				na = search_points[j].numNodesTraversed;
		
                    if(search_points[j].numNodesTraversed > maximum)
                        maximum = search_points[j].numNodesTraversed;
                    all += search_points[j].numNodesTraversed;       
            }

            printf("%d\n", maximum);
            maximum_sum += maximum;
            all_sum += all;
        }
    }	

	printf("avg nodes: %f\n", (float)sum_nodes_traversed / nsearchpoints);

//	sum_nodes_traversed = 0;
//	for (int i = 0; i < nsearchpoints; i++)
//    {
//		sum_nodes_traversed += search_points[i].numNodesTraversed;
//	}
    printf("@ sum_nodes_traversed: %ld\n", sum_nodes_traversed);
	printf("@ avg_nodes_traversed: %f\n", (float)sum_nodes_traversed / nsearchpoints);
	#endif 

	// print results
	if(verbose_flag) {
		for(i = 0; i < nsearchpoints; i++) {
			if(search_points[i].closest >= 0) {
				printf("%d: %d (%2.3f)\n", i, training_points[search_points[i].closest].label, search_points[i].closestDist);
			}
		}
	}
	
	TIME_ELAPSED_PRINT(construct_tree, stdout);
	TIME_ELAPSED_PRINT(gpu_build_tree, stdout);
	TIME_ELAPSED_PRINT(init_kernel, stdout);
	TIME_ELAPSED_PRINT(gpu_copy_to, stdout);
	TIME_ELAPSED_PRINT(kernel, stdout);
	TIME_ELAPSED_PRINT(sort, stdout);
	TIME_ELAPSED_PRINT(gpu_copy_from, stdout);
	TIME_ELAPSED_PRINT(traversal_time, stdout);
	TIME_ELAPSED_PRINT(runtime, stdout);

	return 0;
}

void read_input(int argc, char **argv) {
	unsigned long long i, j, k, c;
	//float min = FLT_MAX;
	//float max = FLT_MIN;
	FILE *in;

	if(argc < 3) {
		fprintf(stderr, "usage: nn [-c] [-v] [-s] <input_file> <npoints> [<nsearchpoints>]\n");
		exit(1);
	}

	while((c = getopt(argc, argv, "cvsrw")) != -1) {
		switch(c) {
		case 'c':
			check_flag = 1;
			break;

		case 'v':
			verbose_flag = 1;
			break;

		case 's':
			sort_flag = 1;
			break;

        case 'r':
            ratio_flag = 1;
            break;

        case 'w':
            warp_flag = 1;
            break;
		
        case '?':
			fprintf(stderr, "Error: unknown option.\n");
			exit(1);
			break;

		default:
			abort();
		}
	}
	
	for(i = optind; i < argc; i++) {
		switch(i - optind) {
		case 0:
			input_file = argv[i];
			break;

		case 1:
				npoints = atoi(argv[i]);
				nsearchpoints = npoints;
				if(npoints <= 0) {
					fprintf(stderr, "Not enough points.\n");
					exit(1);
				}
				break;

		case 2:
			nsearchpoints = atoi(argv[i]);
			if(nsearchpoints <= 0) {
				fprintf(stderr, "Not enough search points.");
				exit(1);
			}
			break;
		}
	}

	training_points = alloc_points(npoints);
	search_points = alloc_points(nsearchpoints);

	if(strcmp(input_file, "random") == 0) {
		for(i = 0; i < npoints; i++) {
			training_points[i].label = i;
			for(j = 0; j < DIM; j++) {
				training_points[i].coord[j] = 1.0 + (float)rand() / RAND_MAX;			
			}
		}

		for(i = 0; i < nsearchpoints; i++) {
			search_points[i].label = npoints + i;
			for(j = 0; j < DIM; j++) {
				search_points[i].coord[j] = 1.0 + (float)rand() / RAND_MAX;			
			}
		}

	} else {
		in = fopen(input_file, "r");
		if(in == NULL) {
			fprintf(stderr, "Could not open %s\n", input_file);
			exit(1);
		}

		for(i = 0; i < npoints; i++) {
			read_point(in, &training_points[i]);
		}

		for(i = 0; i < nsearchpoints; i++) {
			read_point(in, &search_points[i]);
		}

		fclose(in);
	}
}

Point* alloc_points(int n) {
	int i, j;
	Point *points;
	SAFE_MALLOC(points, sizeof(Point) * n);
	for (i = 0; i < n; i++) {
		points[i].closestDist = FLT_MAX;
		points[i].closest = -1;
		#ifdef TRACK_TRAVERSALS
		points[i].numNodesTraversed = 0;
		#endif
	}
	return points;
}

KDCell* alloc_kdcell() {
	int i;
	KDCell *cell;
	SAFE_MALLOC(cell, sizeof(KDCell));
	for (i = 0; i < DIM; i++) {
		cell->min[i] = FLT_MAX;
		cell->max[i] = FLT_MIN;
	}

	for (i = 0; i < MAX_POINTS_IN_CELL; i++) {
		cell->points[i] = -1;
	}

	cell->left = NULL;
	cell->right = NULL;
	return cell;
}

void read_point(FILE *in, Point *p) {
	int j;
	if(fscanf(in, "%d", &p->label) != 1) {
		fprintf(stderr, "Input file not large enough.\n");
		exit(1);
	}
	for(j = 0; j < DIM; j++) {
		if(fscanf(in, "%f", &p->coord[j]) != 1) {
			fprintf(stderr, "Input file not large enough.\n");
			exit(1);
		}
	}
}
